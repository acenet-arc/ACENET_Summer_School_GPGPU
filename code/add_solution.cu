
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// ... define function 'add' ...
__global__ void add(int *da, int *db, int *dc) {
   *dc = *da + *db;
}

int main(int argc, char **argv) {
  int a, b, c;        // We've chosen static allocation here for host storage..
  int *da, *db, *dc;  // ...but device storage must be dynamically allocated
  a = atoi(argv[1]);  // Read the addends from the command line args
  b = atoi(argv[2]);

  // ... manage memory ...
  hipMalloc((void **)&da, sizeof(int));
  hipMalloc((void **)&db, sizeof(int));
  hipMalloc((void **)&dc, sizeof(int));

  // ... move data ...
  hipMemcpy(da, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(db, &b, sizeof(int), hipMemcpyHostToDevice);

  add<<<1,1>>>(da, db, dc);

  // ... move data ...
  hipMemcpy(&c, dc, sizeof(int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  printf("%d + %d -> %d\n", a, b, c);

  // ... manage memory ...
  hipFree(da); hipFree(db); hipFree(dc);
}
