
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *da, int *db, int *dc) {
   dc[threadIdx.x] = da[threadIdx.x] + db[threadIdx.x];
}

int main(int argc, char **argv) {
  int a_in = atoi(argv[1]);       // Read the addends from the command line
  int b_in = atoi(argv[2]);
  int N = atoi(argv[3]);          // Read the length of the vectors
  int gpuThreads = atoi(argv[4]); // Read the number of CUDA threads to use

  int *a, *b, *c;
  int *da, *db, *dc;

  int bytes = N *sizeof(int);
  a = (int *)malloc(bytes);
  b = (int *)malloc(bytes);
  c = (int *)malloc(bytes);
  for (int i=0; i<N; i++) {
     a[i] = a_in; b[i] = b_in; c[i] = 0; }

  hipMalloc((void **)&da, bytes);
  hipMalloc((void **)&db, bytes);
  hipMalloc((void **)&dc, bytes);

  hipMemcpy(da, a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(db, b, bytes, hipMemcpyHostToDevice);

  add<<<1,gpuThreads>>>(da, db, dc);

  hipMemcpy(c, dc, bytes, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  printf("%d + %d -> %d\n", a[0], b[0], c[0]);
  printf(" ...\n");
  printf("%d + %d -> %d\n", a[N-1], b[N-1], c[N-1]);

  hipFree(da); hipFree(db); hipFree(dc);
  free(a); free(b); free(c);
}
